#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <float.h>
#include "mypgm.h"

//kernel: funkcija koju izvtrsavaju niti GPUa

__global__ void sobel_gpu(float* image1, float* image2, int w, int h) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float dx, dy;

    if (x > 0 && y > 0 && x < w - 1 && y < h - 1){


        dy = (image1[(y - 1) * w + (x - 1)]) + (2 * image1[(y - 1) * w + x]) + (image1[(y - 1) * w + (x + 1)]) +
            (-1 * image1[(y + 1) * w + (x - 1)]) + (-2 * image1[(y + 1) * w + x]) + (-1 * image1[(y + 1) * w + (x + 1)]);

        dx = (-1 * image1[(y - 1) * w + (x - 1)]) + (-2 * image1[y * w + (x - 1)]) 
            + (-1 * image1[(y + 1) * w + (x - 1)]) + (image1[(y - 1) * w + (x + 1)]) 
            + (2 * image1[y * w + (x + 1)]) + (image1[(y + 1) * w + (x + 1)]);

       

        image2[y * w + x] = sqrt((dx * dx) + (dy * dy));
    }
}

//host: funkcija koja se izvršava na procesoru

__host__ void sobel_cpu(float image1[1024][1024], float image3[1024][1024], int w, int h) {

    float dx, dy;

    for (int i = 0; i < w-2; i++) {
        for (int j = 0; j < h-2; j++) {

           
                dy = (-1 * image1[i][j]) + (-2 * image1[i][j+1]) + (-1 * image1[i][j+2]) +
                    (1 * image1[i+2][j]) + (2 * image1[i+2][j+1]) + (image1[i+2][j+2]);


                dx = (-1 * image1[i][j]) + (1 * image1[i][j + 2])
                    + (-2* image1[i+1][j]) + (2 * image1[i+1][j+2])
                    + (-1 * image1[i+2][j]) + (1 * image1[i+2][j+2]);

                image3[i][j] = sqrt((dx * dx) + (dy * dy));
        }
    }
}

int main() {

    //Ucitavanje Slike (slika image1 je globalna varijabla radi jednostavnosti)
    load_image_data();

    int i, j;

    float* deviceInputImageData;
    float* deviceOutputImageData;

    //Alociranje memorije na grafickoj kartici
    hipMalloc((void**)&deviceInputImageData, x_size1 * y_size1 * sizeof(float));
    hipMalloc((void**)&deviceOutputImageData, x_size1 * y_size1 * sizeof(float));

    //kopiranje vrijednosti sa glavne memorije na memoriju grafičke kartice
    hipMemcpy(deviceInputImageData, image1, x_size1 * y_size1 * sizeof(float), hipMemcpyHostToDevice);

    //Inicijaliziramo sat da možemo pratiti vrijeme izvrsavanja GPUa
    std::clock_t start_gpu;
    double duration_gpu;
    start_gpu = std::clock();

    printf("                        ----------- GPU -----------\n");

    //Popunjavanje matrice2 (nove slike)//
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    printf("INICIALIZACIJA image2 (SLIKE_GPU)\n");
    x_size2 = x_size1;
    y_size2 = y_size1;
    for (i = 0; i < y_size2; i++) {
        for (j = 0; j < x_size2; j++) {
            image2[i][j] = 0;
        }
    }

    dim3  threadsPerBlock(16, 16);  //1024x1024
    dim3  numBlocks(64, 64);        //16*64 = 1024  /32*32

    printf("\nPOCETAK: Sobel filtra na GPU\n");
    //Funkcija <<<dimGrid, dimBlock>>>
    sobel_gpu<<<numBlocks, threadsPerBlock>>>(deviceInputImageData, deviceOutputImageData, x_size1, y_size1);

    //Završetak sata GPU 
    duration_gpu = (std::clock() - start_gpu) / (double)CLOCKS_PER_SEC;
    std::cout << "\nZAVRSETAK: Sobel filtra na GPU:    " << duration_gpu << "s   -   USPIJEH!" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");

    //kopiranje vrijednosti sa grafičke kartice u glavnu memoriju
    hipMemcpy(image2, deviceOutputImageData, x_size2 * y_size2 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);

    save_image_data_img2();

    //inicijaliziramo sat da možemo pratiti vrijeme izvrsavanja CPUa
    std::clock_t start_cpu;
    double duration_cpu;
    start_cpu = std::clock();

    printf("                        ----------- CPU -----------\n");

    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    printf("INICIALIZACIJA image3 (SLIKE_CPU)\n");
    x_size3 = x_size1;
    y_size3 = y_size1;
    for (i = 0; i < y_size3; i++) {
        for (j = 0; j < x_size3; j++) {
            image3[i][j] = 0;
        }
    }

    printf("\nPOCETAK: Sobel filtra na CPU\n");

    sobel_cpu(image1, image3, x_size3, y_size3);

    ////Završetak sata CPU 
    duration_cpu = (std::clock() - start_cpu) / (double)CLOCKS_PER_SEC;
    std::cout << "\nZAVRSETAK: Sobel filtra na CPU:   " << duration_cpu << "s   -   USPIJEH!" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");

    //Razlika u vremenu izvršavanja:
    double duration;
    double times;

    duration = duration_cpu - duration_gpu;
    times = duration_cpu / duration_gpu * 100;

    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~");
    std::cout << "\nRAZLIKA VREMENA IZVRSAVANJA CPU - GPU:  " << duration << '\n';
    std::cout << "\nGPU je BRZI od CPUa za:  " << times << " %" << '\n';
    printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\n");
    
    save_image_data_img3();

    return 0;
}

